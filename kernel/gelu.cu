
#include <hip/hip_runtime.h>
#include <cmath>

extern "C"
__global__ void gelu_unfused(float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        float sigmoid_val = 1.0 / (1.0 + exp(-1.702 * x));
        output[idx] = x * sigmoid_val;
    }
}

extern "C"
__global__ void gelu_unfused_grad(float* input, float* grad_output, float* grad_input, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        float sigmoid_val = 1.0 / (1.0 + exp(-1.702 * x));
        float pdf = exp(-0.5 * x * x) * (1 / sqrt(2 * M_PI));
        grad_input[idx] = grad_output[idx] * (sigmoid_val + 1.702 * x * pdf);
    }
}

extern "C"
__global__ void gelu_fused(float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        float exp_val = exp(-1.702 * x);
        float sigmoid_val = 1.0 / (1.0 + exp_val);
        output[idx] = x * sigmoid_val;
    }
}

extern "C"
__global__ void gelu_fused_grad(float* input, float* grad_output, float* grad_input, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        float exp_val = exp(-1.702 * x);
        float sigmoid_val = 1.0 / (1.0 + exp_val);
        float pdf = exp(-0.5 * x * x) * (1 / sqrt(2 * M_PI)); 
        grad_input[idx] = grad_output[idx] * (sigmoid_val + 1.702 * x * pdf);
    }
}

